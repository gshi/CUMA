#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
 #include <dlfcn.h>

#define N  16

#define CUERR do {  \
    if (err != hipSuccess){			\
      printf("ERROR: CU call failed (%s) at file %s, line %d\n",\
	     hipGetErrorString(hipGetLastError()), __FILE__, __LINE__); \
      exit(1);								\
    }						\
  }while(0)



int (*__cuInit)(int);
int (*__cuDeviceGetCount)(int*);
int (*__cuDriverGetVersion)(int*);
int (*__cuDeviceGet)(int*, int);
int (*__cuDeviceGetAttribute)(int*, int, int);
int (*__cuDeviceGetName)(char*, int, int);
int (*__cuDeviceTotalMem)(unsigned int*, int);
int (*__cuDeviceComputeCapability)(int*, int*, int);
int (*__cuCtxCreate)(hipCtx_t*, unsigned int, unsigned int);
int (*__cuCtxDestroy)(void*);
int (*__cuMemAlloc)(unsigned int*, unsigned int);
int (*__cuMemFree)(unsigned int);
int (*__cuMemGetInfo)(unsigned int*, unsigned int*);
#define cudalib RTLD_DEFAULT

int testfunc()
{
    hipDeviceptr_t A = 0;
    hipDeviceptr_t B = 0;
    hipDeviceptr_t C = 0;
    hipError_t err;
    hipDevice_t device;
    hipCtx_t context;
    int devid = 0;

    hipError_t (*real_cuMemAlloc)(hipDeviceptr_t* dptr, size_t bytesize);

    __cuDeviceGetCount = (int(*)(int*)) dlsym(cudalib, "hipGetDeviceCount");
    __cuDriverGetVersion = (int(*)(int*)) dlsym( cudalib, "hipDriverGetVersion" );
    __cuInit = (int(*)(int)) dlsym( cudalib, "hipInit" );
    __cuDeviceGet = (int(*)(int*, int)) dlsym( cudalib, "hipDeviceGet" );
    __cuDeviceGetAttribute = (int(*)(int*, int, int)) dlsym( cudalib, "hipDeviceGetAttribute" );
    __cuDeviceGetName = (int(*)(char*, int, int)) dlsym( cudalib, "hipDeviceGetName" );
    __cuDeviceTotalMem = (int(*)(unsigned int*, int)) dlsym( cudalib, "hipDeviceTotalMem" );
    __cuDeviceComputeCapability = (int(*)(int*, int*, int)) dlsym( cudalib, "hipDeviceComputeCapability" );
    __cuCtxCreate = (int(*)(hipCtx_t*, unsigned int, unsigned int)) dlsym( cudalib, "hipCtxCreate" );
    __cuCtxDestroy = (int(*)(void*)) dlsym( cudalib, "hipCtxDestroy" );
    __cuMemAlloc = (int(*)(unsigned int*, unsigned int)) dlsym( cudalib, "hipMalloc" );
    __cuMemFree = (int(*)(unsigned int)) dlsym( cudalib, "hipFree" );
    __cuMemGetInfo = (int(*)(unsigned int*, unsigned int*)) dlsym( cudalib, "hipMemGetInfo" );




    
    //cuInit(0);
    //err = cuDeviceGet(&device, devid); CUERR;
    //err = cuCtxCreate(&context, 0, device); CUERR;

    (*__cuInit)(0);
    err = (hipError_t)(*__cuDeviceGet)(&device, devid); CUERR;
    err = (hipError_t)(*__cuCtxCreate)(&context, 0, device); CUERR;


    real_cuMemAlloc = (hipError_t (*)(hipDeviceptr_t* dptr, size_t bytesize))
      dlsym(RTLD_NEXT, "hipMalloc");
    printf("real_cuMemAlloc=%p\n", real_cuMemAlloc);
    err = (*real_cuMemAlloc)(&A, sizeof(float)*N); //CUERR;
    printf("err=%d\n", err);
    printf("A=%p\n", A);
    
    //err = cuMemAlloc(&A, sizeof(float)*N); CUERR;
    err = hipMalloc(&B, sizeof(float)*N); //CUERR;
    //err = cuMemAlloc(&C, sizeof(float)*N); CUERR;
    
    printf("B=%p, err=%d\n", B,err);
    printf("C=%p\n", C);
    
    //cuMemFree(A);
    //cuMemFree(B);
    //cuMemFree(C);
    
    //err = cuCtxDetach(context); CUERR;
    return 0;
    
}

int main()
{
  testfunc();
  
  return 0;
}
